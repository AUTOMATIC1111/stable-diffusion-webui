#include "hip/hip_runtime.h"
/*!
**************************************************************************************************
* Deformable DETR
* Copyright (c) 2020 SenseTime. All Rights Reserved.
* Licensed under the Apache License, Version 2.0 [see LICENSE for details]
**************************************************************************************************
* Modified from https://github.com/chengdazhi/Deformable-Convolution-V2-PyTorch/tree/pytorch_1.0.0
**************************************************************************************************
*/

/*!
* Copyright (c) Facebook, Inc. and its affiliates.
* Modified by Bowen Cheng from https://github.com/fundamentalvision/Deformable-DETR
*/

#include <vector>
#include "cuda/ms_deform_im2col_cuda.cuh"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


at::Tensor ms_deform_attn_cuda_forward(
    const at::Tensor &value, 
    const at::Tensor &spatial_shapes,
    const at::Tensor &level_start_index,
    const at::Tensor &sampling_loc,
    const at::Tensor &attn_weight,
    const int im2col_step)
{
    AT_ASSERTM(value.is_contiguous(), "value tensor has to be contiguous");
    AT_ASSERTM(spatial_shapes.is_contiguous(), "spatial_shapes tensor has to be contiguous");
    AT_ASSERTM(level_start_index.is_contiguous(), "level_start_index tensor has to be contiguous");
    AT_ASSERTM(sampling_loc.is_contiguous(), "sampling_loc tensor has to be contiguous");
    AT_ASSERTM(attn_weight.is_contiguous(), "attn_weight tensor has to be contiguous");

    AT_ASSERTM(value.type().is_cuda(), "value must be a CUDA tensor");
    AT_ASSERTM(spatial_shapes.type().is_cuda(), "spatial_shapes must be a CUDA tensor");
    AT_ASSERTM(level_start_index.type().is_cuda(), "level_start_index must be a CUDA tensor");
    AT_ASSERTM(sampling_loc.type().is_cuda(), "sampling_loc must be a CUDA tensor");
    AT_ASSERTM(attn_weight.type().is_cuda(), "attn_weight must be a CUDA tensor");

    const int batch = value.size(0);
    const int spatial_size = value.size(1);
    const int num_heads = value.size(2);
    const int channels = value.size(3);

    const int num_levels = spatial_shapes.size(0);

    const int num_query = sampling_loc.size(1);
    const int num_point = sampling_loc.size(4);

    const int im2col_step_ = std::min(batch, im2col_step);

    AT_ASSERTM(batch % im2col_step_ == 0, "batch(%d) must divide im2col_step(%d)", batch, im2col_step_);
    
    auto output = at::zeros({batch, num_query, num_heads, channels}, value.options());

    const int batch_n = im2col_step_;
    auto output_n = output.view({batch/im2col_step_, batch_n, num_query, num_heads, channels});
    auto per_value_size = spatial_size * num_heads * channels;
    auto per_sample_loc_size = num_query * num_heads * num_levels * num_point * 2;
    auto per_attn_weight_size = num_query * num_heads * num_levels * num_point;
    for (int n = 0; n < batch/im2col_step_; ++n)
    {
        auto columns = output_n.select(0, n);
        AT_DISPATCH_FLOATING_TYPES(value.type(), "ms_deform_attn_forward_cuda", ([&] {
            ms_deformable_im2col_cuda(at::cuda::getCurrentCUDAStream(),
                value.data<scalar_t>() + n * im2col_step_ * per_value_size,
                spatial_shapes.data<int64_t>(),
                level_start_index.data<int64_t>(),
                sampling_loc.data<scalar_t>() + n * im2col_step_ * per_sample_loc_size,
                attn_weight.data<scalar_t>() + n * im2col_step_ * per_attn_weight_size,
                batch_n, spatial_size, num_heads, channels, num_levels, num_query, num_point,
                columns.data<scalar_t>());

        }));
    }

    output = output.view({batch, num_query, num_heads*channels});

    return output;
}


std::vector<at::Tensor> ms_deform_attn_cuda_backward(
    const at::Tensor &value, 
    const at::Tensor &spatial_shapes,
    const at::Tensor &level_start_index,
    const at::Tensor &sampling_loc,
    const at::Tensor &attn_weight,
    const at::Tensor &grad_output,
    const int im2col_step)
{

    AT_ASSERTM(value.is_contiguous(), "value tensor has to be contiguous");
    AT_ASSERTM(spatial_shapes.is_contiguous(), "spatial_shapes tensor has to be contiguous");
    AT_ASSERTM(level_start_index.is_contiguous(), "level_start_index tensor has to be contiguous");
    AT_ASSERTM(sampling_loc.is_contiguous(), "sampling_loc tensor has to be contiguous");
    AT_ASSERTM(attn_weight.is_contiguous(), "attn_weight tensor has to be contiguous");
    AT_ASSERTM(grad_output.is_contiguous(), "grad_output tensor has to be contiguous");

    AT_ASSERTM(value.type().is_cuda(), "value must be a CUDA tensor");
    AT_ASSERTM(spatial_shapes.type().is_cuda(), "spatial_shapes must be a CUDA tensor");
    AT_ASSERTM(level_start_index.type().is_cuda(), "level_start_index must be a CUDA tensor");
    AT_ASSERTM(sampling_loc.type().is_cuda(), "sampling_loc must be a CUDA tensor");
    AT_ASSERTM(attn_weight.type().is_cuda(), "attn_weight must be a CUDA tensor");
    AT_ASSERTM(grad_output.type().is_cuda(), "grad_output must be a CUDA tensor");

    const int batch = value.size(0);
    const int spatial_size = value.size(1);
    const int num_heads = value.size(2);
    const int channels = value.size(3);

    const int num_levels = spatial_shapes.size(0);

    const int num_query = sampling_loc.size(1);
    const int num_point = sampling_loc.size(4);

    const int im2col_step_ = std::min(batch, im2col_step);

    AT_ASSERTM(batch % im2col_step_ == 0, "batch(%d) must divide im2col_step(%d)", batch, im2col_step_);

    auto grad_value = at::zeros_like(value);
    auto grad_sampling_loc = at::zeros_like(sampling_loc);
    auto grad_attn_weight = at::zeros_like(attn_weight);

    const int batch_n = im2col_step_;
    auto per_value_size = spatial_size * num_heads * channels;
    auto per_sample_loc_size = num_query * num_heads * num_levels * num_point * 2;
    auto per_attn_weight_size = num_query * num_heads * num_levels * num_point;
    auto grad_output_n = grad_output.view({batch/im2col_step_, batch_n, num_query, num_heads, channels});
    
    for (int n = 0; n < batch/im2col_step_; ++n)
    {
        auto grad_output_g = grad_output_n.select(0, n);
        AT_DISPATCH_FLOATING_TYPES(value.type(), "ms_deform_attn_backward_cuda", ([&] {
            ms_deformable_col2im_cuda(at::cuda::getCurrentCUDAStream(),
                                    grad_output_g.data<scalar_t>(),
                                    value.data<scalar_t>() + n * im2col_step_ * per_value_size,
                                    spatial_shapes.data<int64_t>(),
                                    level_start_index.data<int64_t>(),
                                    sampling_loc.data<scalar_t>() + n * im2col_step_ * per_sample_loc_size,
                                    attn_weight.data<scalar_t>() + n * im2col_step_ * per_attn_weight_size,
                                    batch_n, spatial_size, num_heads, channels, num_levels, num_query, num_point,
                                    grad_value.data<scalar_t>() +  n * im2col_step_ * per_value_size,
                                    grad_sampling_loc.data<scalar_t>() + n * im2col_step_ * per_sample_loc_size,
                                    grad_attn_weight.data<scalar_t>() + n * im2col_step_ * per_attn_weight_size);

        }));
    }

    return {
        grad_value, grad_sampling_loc, grad_attn_weight
    };
}