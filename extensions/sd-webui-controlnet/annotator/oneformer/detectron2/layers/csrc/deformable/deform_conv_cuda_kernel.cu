#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.

// modified from
// https://github.com/open-mmlab/mmdetection/blob/master/mmdet/ops/dcn/src/deform_conv_cuda_kernel.cu
// Original license: Apache 2.0
// clang-format off

// modify from
// https://github.com/chengdazhi/Deformable-Convolution-V2-PyTorch/blob/mmdetection/mmdet/ops/dcn/src/deform_conv_cuda_kernel.cu

/*!
 ******************* BEGIN Caffe Copyright Notice and Disclaimer *****************
 *
 * COPYRIGHT
 *
 * All contributions by the University of California:
 * Copyright (c) 2014-2017 The Regents of the University of California (Regents)
 * All rights reserved.
 *
 * All other contributions:
 * Copyright (c) 2014-2017, the respective contributors
 * All rights reserved.
 *
 * Caffe uses a shared copyright model: each contributor holds copyright over
 * their contributions to Caffe. The project versioning records all such
 * contribution and copyright details. If a contributor wants to further mark
 * their specific copyright on a particular contribution, they should indicate
 * their copyright solely in the commit message of the change when it is
 * committed.
 *
 * LICENSE
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 *AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
 *FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 *DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 *SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 *OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 *OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * CONTRIBUTION AGREEMENT
 *
 * By contributing to the BVLC/caffe repository through pull-request, comment,
 * or otherwise, the contributor releases their content to the
 * license and copyright terms herein.
 *
 ***************** END Caffe Copyright Notice and Disclaimer *********************
 *
 * Copyright (c) 2018 Microsoft
 * Licensed under The MIT License [see LICENSE for details]
 * \file modulated_deformable_im2col.cuh
 * \brief Function definitions of converting an image to
 * column matrix based on kernel, padding, dilation, and offset.
 * These functions are mainly used in deformable convolution operators.
 * \ref: https://arxiv.org/abs/1703.06211
 * \author Yuwen Xiong, Haozhi Qi, Jifeng Dai, Xizhou Zhu, Han Hu, Dazhi Cheng
 */

#include <ATen/ATen.h>
#include <c10/cuda/CUDAGuard.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <THC/THCAtomics.cuh>

using namespace at;

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)


namespace {

const int CUDA_NUM_THREADS = 1024;
const int kMaxGridNum = 65535;

inline int GET_BLOCKS(const int N) {
  return std::min(kMaxGridNum, (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS);
}

}

template <typename scalar_t>
__device__ scalar_t deformable_im2col_bilinear(
    const scalar_t* bottom_data,
    const int data_width,
    const int height,
    const int width,
    scalar_t h,
    scalar_t w) {
  int h_low = floor(h);
  int w_low = floor(w);
  int h_high = h_low + 1;
  int w_high = w_low + 1;

  scalar_t lh = h - h_low;
  scalar_t lw = w - w_low;
  scalar_t hh = 1 - lh, hw = 1 - lw;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0)
    v1 = bottom_data[h_low * data_width + w_low];
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
    v2 = bottom_data[h_low * data_width + w_high];
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
    v3 = bottom_data[h_high * data_width + w_low];
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
    v4 = bottom_data[h_high * data_width + w_high];

  scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename scalar_t>
__device__ scalar_t get_gradient_weight(
    scalar_t argmax_h,
    scalar_t argmax_w,
    const int h,
    const int w,
    const int height,
    const int width) {
  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 ||
      argmax_w >= width) {
    // empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  scalar_t weight = 0;
  if (h == argmax_h_low && w == argmax_w_low)
    weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
  if (h == argmax_h_low && w == argmax_w_high)
    weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
  if (h == argmax_h_high && w == argmax_w_low)
    weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
  if (h == argmax_h_high && w == argmax_w_high)
    weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
  return weight;
}

template <typename scalar_t>
__device__ scalar_t get_coordinate_weight(
    scalar_t argmax_h,
    scalar_t argmax_w,
    const int height,
    const int width,
    const scalar_t* im_data,
    const int data_width,
    const int bp_dir) {
  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 ||
      argmax_w >= width) {
    // empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  scalar_t weight = 0;

  if (bp_dir == 0) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_w_low + 1 - argmax_w) *
          im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += -1 * (argmax_w - argmax_w_low) *
          im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += (argmax_w_low + 1 - argmax_w) *
          im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_w - argmax_w_low) *
          im_data[argmax_h_high * data_width + argmax_w_high];
  } else if (bp_dir == 1) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_h_low + 1 - argmax_h) *
          im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += (argmax_h_low + 1 - argmax_h) *
          im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += -1 * (argmax_h - argmax_h_low) *
          im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_h - argmax_h_low) *
          im_data[argmax_h_high * data_width + argmax_w_high];
  }

  return weight;
}

template <typename scalar_t>
__global__ void deformable_im2col_gpu_kernel(
    const int n,
    const scalar_t* data_im,
    const scalar_t* data_offset,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int batch_size,
    const int num_channels,
    const int deformable_group,
    const int height_col,
    const int width_col,
    scalar_t* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    // index index of output matrix
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int b_col = (index / width_col / height_col) % batch_size;
    const int c_im = (index / width_col / height_col) / batch_size;
    const int c_col = c_im * kernel_h * kernel_w;

    // compute deformable group index
    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;
    scalar_t* data_col_ptr = data_col +
        ((c_col * batch_size + b_col) * height_col + h_col) * width_col + w_col;
    // const scalar_t* data_im_ptr = data_im + ((b_col * num_channels + c_im) *
    // height + h_in) * width + w_in;
    const scalar_t* data_im_ptr =
        data_im + (b_col * num_channels + c_im) * height * width;
    const scalar_t* data_offset_ptr = data_offset +
        (b_col * deformable_group + deformable_group_index) * 2 * kernel_h *
            kernel_w * height_col * width_col;

    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        const int data_offset_h_ptr =
            ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr =
            ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col +
            w_col;
        const scalar_t offset_h = data_offset_ptr[data_offset_h_ptr];
        const scalar_t offset_w = data_offset_ptr[data_offset_w_ptr];
        scalar_t val = static_cast<scalar_t>(0);
        const scalar_t h_im = h_in + i * dilation_h + offset_h;
        const scalar_t w_im = w_in + j * dilation_w + offset_w;
        if (h_im > -1 && w_im > -1 && h_im < height && w_im < width) {
          // const scalar_t map_h = i * dilation_h + offset_h;
          // const scalar_t map_w = j * dilation_w + offset_w;
          // const int cur_height = height - h_in;
          // const int cur_width = width - w_in;
          // val = deformable_im2col_bilinear(data_im_ptr, width, cur_height,
          // cur_width, map_h, map_w);
          val = deformable_im2col_bilinear(
              data_im_ptr, width, height, width, h_im, w_im);
        }
        *data_col_ptr = val;
        data_col_ptr += batch_size * height_col * width_col;
      }
    }
  }
}


template <typename scalar_t>
__global__ void deformable_col2im_gpu_kernel(
    const int n,
    const scalar_t* data_col,
    const scalar_t* data_offset,
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int batch_size,
    const int deformable_group,
    const int height_col,
    const int width_col,
    scalar_t* grad_im) {
  CUDA_KERNEL_LOOP(index, n) {
    const int j = (index / width_col / height_col / batch_size) % kernel_w;
    const int i =
        (index / width_col / height_col / batch_size / kernel_w) % kernel_h;
    const int c =
        index / width_col / height_col / batch_size / kernel_w / kernel_h;
    // compute the start and end of the output

    const int deformable_group_index = c / channel_per_deformable_group;

    int w_out = index % width_col;
    int h_out = (index / width_col) % height_col;
    int b = (index / width_col / height_col) % batch_size;
    int w_in = w_out * stride_w - pad_w;
    int h_in = h_out * stride_h - pad_h;

    const scalar_t* data_offset_ptr = data_offset +
        (b * deformable_group + deformable_group_index) * 2 * kernel_h *
            kernel_w * height_col * width_col;
    const int data_offset_h_ptr =
        ((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out;
    const int data_offset_w_ptr =
        ((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col + w_out;
    const scalar_t offset_h = data_offset_ptr[data_offset_h_ptr];
    const scalar_t offset_w = data_offset_ptr[data_offset_w_ptr];
    const scalar_t cur_inv_h_data = h_in + i * dilation_h + offset_h;
    const scalar_t cur_inv_w_data = w_in + j * dilation_w + offset_w;

    const scalar_t cur_top_grad = data_col[index];
    const int cur_h = (int)cur_inv_h_data;
    const int cur_w = (int)cur_inv_w_data;
    for (int dy = -2; dy <= 2; dy++) {
      for (int dx = -2; dx <= 2; dx++) {
        if (cur_h + dy >= 0 && cur_h + dy < height && cur_w + dx >= 0 &&
            cur_w + dx < width && abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            abs(cur_inv_w_data - (cur_w + dx)) < 1) {
          int cur_bottom_grad_pos =
              ((b * channels + c) * height + cur_h + dy) * width + cur_w + dx;
          scalar_t weight = get_gradient_weight(
              cur_inv_h_data,
              cur_inv_w_data,
              cur_h + dy,
              cur_w + dx,
              height,
              width);
          atomicAdd(grad_im + cur_bottom_grad_pos, weight * cur_top_grad);
        }
      }
    }
  }
}


template <typename scalar_t>
__global__ void deformable_col2im_coord_gpu_kernel(
    const int n,
    const scalar_t* data_col,
    const scalar_t* data_im,
    const scalar_t* data_offset,
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int batch_size,
    const int offset_channels,
    const int deformable_group,
    const int height_col,
    const int width_col,
    scalar_t* grad_offset) {
  CUDA_KERNEL_LOOP(index, n) {
    scalar_t val = 0;
    int w = index % width_col;
    int h = (index / width_col) % height_col;
    int c = (index / width_col / height_col) % offset_channels;
    int b = (index / width_col / height_col) / offset_channels;
    // compute the start and end of the output

    const int deformable_group_index = c / (2 * kernel_h * kernel_w);
    const int col_step = kernel_h * kernel_w;
    int cnt = 0;
    const scalar_t* data_col_ptr = data_col +
        deformable_group_index * channel_per_deformable_group * batch_size *
            width_col * height_col;
    const scalar_t* data_im_ptr = data_im +
        (b * deformable_group + deformable_group_index) *
            channel_per_deformable_group / kernel_h / kernel_w * height * width;
    const scalar_t* data_offset_ptr = data_offset +
        (b * deformable_group + deformable_group_index) * 2 * kernel_h *
            kernel_w * height_col * width_col;

    const int offset_c = c - deformable_group_index * 2 * kernel_h * kernel_w;

    for (int col_c = (offset_c / 2); col_c < channel_per_deformable_group;
         col_c += col_step) {
      const int col_pos =
          (((col_c * batch_size + b) * height_col) + h) * width_col + w;
      const int bp_dir = offset_c % 2;

      int j = (col_pos / width_col / height_col / batch_size) % kernel_w;
      int i =
          (col_pos / width_col / height_col / batch_size / kernel_w) % kernel_h;
      int w_out = col_pos % width_col;
      int h_out = (col_pos / width_col) % height_col;
      int w_in = w_out * stride_w - pad_w;
      int h_in = h_out * stride_h - pad_h;
      const int data_offset_h_ptr =
          (((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out);
      const int data_offset_w_ptr =
          (((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col +
           w_out);
      const scalar_t offset_h = data_offset_ptr[data_offset_h_ptr];
      const scalar_t offset_w = data_offset_ptr[data_offset_w_ptr];
      scalar_t inv_h = h_in + i * dilation_h + offset_h;
      scalar_t inv_w = w_in + j * dilation_w + offset_w;
      if (inv_h <= -1 || inv_w <= -1 || inv_h >= height || inv_w >= width) {
        inv_h = inv_w = -2;
      }
      const scalar_t weight = get_coordinate_weight(
          inv_h,
          inv_w,
          height,
          width,
          data_im_ptr + cnt * height * width,
          width,
          bp_dir);
      val += weight * data_col_ptr[col_pos];
      cnt += 1;
    }

    grad_offset[index] = val;
  }
}


namespace detectron2 {

void deformable_im2col(
    const at::Tensor data_im,
    const at::Tensor data_offset,
    const int channels,
    const int height,
    const int width,
    const int ksize_h,
    const int ksize_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int parallel_imgs,
    const int deformable_group,
    at::Tensor data_col) {
  // num_axes should be smaller than block size
  // todo: check parallel_imgs is correctly passed in
  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * height_col * width_col * parallel_imgs;
  int channel_per_deformable_group = channels / deformable_group;

  at::cuda::CUDAGuard device_guard(data_im.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_im.scalar_type(), "deformable_im2col_gpu", ([&] {
        const scalar_t* data_im_ = data_im.data_ptr<scalar_t>();
        const scalar_t* data_offset_ = data_offset.data_ptr<scalar_t>();
        scalar_t* data_col_ = data_col.data_ptr<scalar_t>();

        deformable_im2col_gpu_kernel<<<
            GET_BLOCKS(num_kernels),
            CUDA_NUM_THREADS,
            0,
            stream>>>(
            num_kernels,
            data_im_,
            data_offset_,
            height,
            width,
            ksize_h,
            ksize_w,
            pad_h,
            pad_w,
            stride_h,
            stride_w,
            dilation_h,
            dilation_w,
            channel_per_deformable_group,
            parallel_imgs,
            channels,
            deformable_group,
            height_col,
            width_col,
            data_col_);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in deformable_im2col: %s\n", hipGetErrorString(err));
  }
}


void deformable_col2im(
    const at::Tensor data_col,
    const at::Tensor data_offset,
    const int channels,
    const int height,
    const int width,
    const int ksize_h,
    const int ksize_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int parallel_imgs,
    const int deformable_group,
    at::Tensor grad_im) {
  // todo: make sure parallel_imgs is passed in correctly
  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels =
      channels * ksize_h * ksize_w * height_col * width_col * parallel_imgs;
  int channel_per_deformable_group = channels / deformable_group;

  at::cuda::CUDAGuard device_guard(data_col.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "deformable_col2im_gpu", ([&] {
        const scalar_t* data_col_ = data_col.data_ptr<scalar_t>();
        const scalar_t* data_offset_ = data_offset.data_ptr<scalar_t>();
        scalar_t* grad_im_ = grad_im.data_ptr<scalar_t>();

        deformable_col2im_gpu_kernel<<<
            GET_BLOCKS(num_kernels),
            CUDA_NUM_THREADS,
            0,
            stream>>>(
            num_kernels,
            data_col_,
            data_offset_,
            channels,
            height,
            width,
            ksize_h,
            ksize_w,
            pad_h,
            pad_w,
            stride_h,
            stride_w,
            dilation_h,
            dilation_w,
            channel_per_deformable_group,
            parallel_imgs,
            deformable_group,
            height_col,
            width_col,
            grad_im_);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in deformable_col2im: %s\n", hipGetErrorString(err));
  }
}


void deformable_col2im_coord(
    const at::Tensor data_col,
    const at::Tensor data_im,
    const at::Tensor data_offset,
    const int channels,
    const int height,
    const int width,
    const int ksize_h,
    const int ksize_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int parallel_imgs,
    const int deformable_group,
    at::Tensor grad_offset) {
  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = height_col * width_col * 2 * ksize_h * ksize_w *
      deformable_group * parallel_imgs;
  int channel_per_deformable_group =
      channels * ksize_h * ksize_w / deformable_group;

  at::cuda::CUDAGuard device_guard(data_col.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "deformable_col2im_coord_gpu", ([&] {
        const scalar_t* data_col_ = data_col.data_ptr<scalar_t>();
        const scalar_t* data_im_ = data_im.data_ptr<scalar_t>();
        const scalar_t* data_offset_ = data_offset.data_ptr<scalar_t>();
        scalar_t* grad_offset_ = grad_offset.data_ptr<scalar_t>();

        deformable_col2im_coord_gpu_kernel<<<
            GET_BLOCKS(num_kernels),
            CUDA_NUM_THREADS,
            0,
            stream>>>(
            num_kernels,
            data_col_,
            data_im_,
            data_offset_,
            channels,
            height,
            width,
            ksize_h,
            ksize_w,
            pad_h,
            pad_w,
            stride_h,
            stride_w,
            dilation_h,
            dilation_w,
            channel_per_deformable_group,
            parallel_imgs,
            2 * ksize_h * ksize_w * deformable_group,
            deformable_group,
            height_col,
            width_col,
            grad_offset_);
      }));
}

} // namespace detectron2


template <typename scalar_t>
__device__ scalar_t dmcn_im2col_bilinear(
    const scalar_t* bottom_data,
    const int data_width,
    const int height,
    const int width,
    scalar_t h,
    scalar_t w) {
  int h_low = floor(h);
  int w_low = floor(w);
  int h_high = h_low + 1;
  int w_high = w_low + 1;

  scalar_t lh = h - h_low;
  scalar_t lw = w - w_low;
  scalar_t hh = 1 - lh, hw = 1 - lw;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0)
    v1 = bottom_data[h_low * data_width + w_low];
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
    v2 = bottom_data[h_low * data_width + w_high];
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
    v3 = bottom_data[h_high * data_width + w_low];
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
    v4 = bottom_data[h_high * data_width + w_high];

  scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename scalar_t>
__device__ scalar_t dmcn_get_gradient_weight(
    scalar_t argmax_h,
    scalar_t argmax_w,
    const int h,
    const int w,
    const int height,
    const int width) {
  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 ||
      argmax_w >= width) {
    // empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  scalar_t weight = 0;
  if (h == argmax_h_low && w == argmax_w_low)
    weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
  if (h == argmax_h_low && w == argmax_w_high)
    weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
  if (h == argmax_h_high && w == argmax_w_low)
    weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
  if (h == argmax_h_high && w == argmax_w_high)
    weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
  return weight;
}

template <typename scalar_t>
__device__ scalar_t dmcn_get_coordinate_weight(
    scalar_t argmax_h,
    scalar_t argmax_w,
    const int height,
    const int width,
    const scalar_t* im_data,
    const int data_width,
    const int bp_dir) {
  if (argmax_h <= -1 || argmax_h >= height || argmax_w <= -1 ||
      argmax_w >= width) {
    // empty
    return 0;
  }

  int argmax_h_low = floor(argmax_h);
  int argmax_w_low = floor(argmax_w);
  int argmax_h_high = argmax_h_low + 1;
  int argmax_w_high = argmax_w_low + 1;

  scalar_t weight = 0;

  if (bp_dir == 0) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_w_low + 1 - argmax_w) *
          im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += -1 * (argmax_w - argmax_w_low) *
          im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += (argmax_w_low + 1 - argmax_w) *
          im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_w - argmax_w_low) *
          im_data[argmax_h_high * data_width + argmax_w_high];
  } else if (bp_dir == 1) {
    if (argmax_h_low >= 0 && argmax_w_low >= 0)
      weight += -1 * (argmax_h_low + 1 - argmax_h) *
          im_data[argmax_h_low * data_width + argmax_w_low];
    if (argmax_h_low >= 0 && argmax_w_high <= width - 1)
      weight += (argmax_h_low + 1 - argmax_h) *
          im_data[argmax_h_low * data_width + argmax_w_high];
    if (argmax_h_high <= height - 1 && argmax_w_low >= 0)
      weight += -1 * (argmax_h - argmax_h_low) *
          im_data[argmax_h_high * data_width + argmax_w_low];
    if (argmax_h_high <= height - 1 && argmax_w_high <= width - 1)
      weight += (argmax_h - argmax_h_low) *
          im_data[argmax_h_high * data_width + argmax_w_high];
  }

  return weight;
}

template <typename scalar_t>
__global__ void modulated_deformable_im2col_gpu_kernel(
    const int n,
    const scalar_t* data_im,
    const scalar_t* data_offset,
    const scalar_t* data_mask,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int batch_size,
    const int num_channels,
    const int deformable_group,
    const int height_col,
    const int width_col,
    scalar_t* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    // index index of output matrix
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int b_col = (index / width_col / height_col) % batch_size;
    const int c_im = (index / width_col / height_col) / batch_size;
    const int c_col = c_im * kernel_h * kernel_w;

    // compute deformable group index
    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;

    scalar_t* data_col_ptr = data_col +
        ((c_col * batch_size + b_col) * height_col + h_col) * width_col + w_col;
    // const float* data_im_ptr = data_im + ((b_col * num_channels + c_im) *
    // height + h_in) * width + w_in;
    const scalar_t* data_im_ptr =
        data_im + (b_col * num_channels + c_im) * height * width;
    const scalar_t* data_offset_ptr = data_offset +
        (b_col * deformable_group + deformable_group_index) * 2 * kernel_h *
            kernel_w * height_col * width_col;

    const scalar_t* data_mask_ptr = data_mask +
        (b_col * deformable_group + deformable_group_index) * kernel_h *
            kernel_w * height_col * width_col;

    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        const int data_offset_h_ptr =
            ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr =
            ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col +
            w_col;
        const int data_mask_hw_ptr =
            ((i * kernel_w + j) * height_col + h_col) * width_col + w_col;
        const scalar_t offset_h = data_offset_ptr[data_offset_h_ptr];
        const scalar_t offset_w = data_offset_ptr[data_offset_w_ptr];
        const scalar_t mask = data_mask_ptr[data_mask_hw_ptr];
        scalar_t val = static_cast<scalar_t>(0);
        const scalar_t h_im = h_in + i * dilation_h + offset_h;
        const scalar_t w_im = w_in + j * dilation_w + offset_w;
        // if (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) {
        if (h_im > -1 && w_im > -1 && h_im < height && w_im < width) {
          // const float map_h = i * dilation_h + offset_h;
          // const float map_w = j * dilation_w + offset_w;
          // const int cur_height = height - h_in;
          // const int cur_width = width - w_in;
          // val = dmcn_im2col_bilinear(data_im_ptr, width, cur_height,
          // cur_width, map_h, map_w);
          val = dmcn_im2col_bilinear(
              data_im_ptr, width, height, width, h_im, w_im);
        }
        *data_col_ptr = val * mask;
        data_col_ptr += batch_size * height_col * width_col;
        // data_col_ptr += height_col * width_col;
      }
    }
  }
}

template <typename scalar_t>
__global__ void modulated_deformable_col2im_gpu_kernel(
    const int n,
    const scalar_t* data_col,
    const scalar_t* data_offset,
    const scalar_t* data_mask,
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int batch_size,
    const int deformable_group,
    const int height_col,
    const int width_col,
    scalar_t* grad_im) {
  CUDA_KERNEL_LOOP(index, n) {
    const int j = (index / width_col / height_col / batch_size) % kernel_w;
    const int i =
        (index / width_col / height_col / batch_size / kernel_w) % kernel_h;
    const int c =
        index / width_col / height_col / batch_size / kernel_w / kernel_h;
    // compute the start and end of the output

    const int deformable_group_index = c / channel_per_deformable_group;

    int w_out = index % width_col;
    int h_out = (index / width_col) % height_col;
    int b = (index / width_col / height_col) % batch_size;
    int w_in = w_out * stride_w - pad_w;
    int h_in = h_out * stride_h - pad_h;

    const scalar_t* data_offset_ptr = data_offset +
        (b * deformable_group + deformable_group_index) * 2 * kernel_h *
            kernel_w * height_col * width_col;
    const scalar_t* data_mask_ptr = data_mask +
        (b * deformable_group + deformable_group_index) * kernel_h * kernel_w *
            height_col * width_col;
    const int data_offset_h_ptr =
        ((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out;
    const int data_offset_w_ptr =
        ((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col + w_out;
    const int data_mask_hw_ptr =
        ((i * kernel_w + j) * height_col + h_out) * width_col + w_out;
    const scalar_t offset_h = data_offset_ptr[data_offset_h_ptr];
    const scalar_t offset_w = data_offset_ptr[data_offset_w_ptr];
    const scalar_t mask = data_mask_ptr[data_mask_hw_ptr];
    const scalar_t cur_inv_h_data = h_in + i * dilation_h + offset_h;
    const scalar_t cur_inv_w_data = w_in + j * dilation_w + offset_w;

    const scalar_t cur_top_grad = data_col[index] * mask;
    const int cur_h = (int)cur_inv_h_data;
    const int cur_w = (int)cur_inv_w_data;
    for (int dy = -2; dy <= 2; dy++) {
      for (int dx = -2; dx <= 2; dx++) {
        if (cur_h + dy >= 0 && cur_h + dy < height && cur_w + dx >= 0 &&
            cur_w + dx < width && abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            abs(cur_inv_w_data - (cur_w + dx)) < 1) {
          int cur_bottom_grad_pos =
              ((b * channels + c) * height + cur_h + dy) * width + cur_w + dx;
          scalar_t weight = dmcn_get_gradient_weight(
              cur_inv_h_data,
              cur_inv_w_data,
              cur_h + dy,
              cur_w + dx,
              height,
              width);
          atomicAdd(grad_im + cur_bottom_grad_pos, weight * cur_top_grad);
        }
      }
    }
  }
}

template <typename scalar_t>
__global__ void modulated_deformable_col2im_coord_gpu_kernel(
    const int n,
    const scalar_t* data_col,
    const scalar_t* data_im,
    const scalar_t* data_offset,
    const scalar_t* data_mask,
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int batch_size,
    const int offset_channels,
    const int deformable_group,
    const int height_col,
    const int width_col,
    scalar_t* grad_offset,
    scalar_t* grad_mask) {
  CUDA_KERNEL_LOOP(index, n) {
    scalar_t val = 0, mval = 0;
    int w = index % width_col;
    int h = (index / width_col) % height_col;
    int c = (index / width_col / height_col) % offset_channels;
    int b = (index / width_col / height_col) / offset_channels;
    // compute the start and end of the output

    const int deformable_group_index = c / (2 * kernel_h * kernel_w);
    const int col_step = kernel_h * kernel_w;
    int cnt = 0;
    const scalar_t* data_col_ptr = data_col +
        deformable_group_index * channel_per_deformable_group * batch_size *
            width_col * height_col;
    const scalar_t* data_im_ptr = data_im +
        (b * deformable_group + deformable_group_index) *
            channel_per_deformable_group / kernel_h / kernel_w * height * width;
    const scalar_t* data_offset_ptr = data_offset +
        (b * deformable_group + deformable_group_index) * 2 * kernel_h *
            kernel_w * height_col * width_col;
    const scalar_t* data_mask_ptr = data_mask +
        (b * deformable_group + deformable_group_index) * kernel_h * kernel_w *
            height_col * width_col;

    const int offset_c = c - deformable_group_index * 2 * kernel_h * kernel_w;

    for (int col_c = (offset_c / 2); col_c < channel_per_deformable_group;
         col_c += col_step) {
      const int col_pos =
          (((col_c * batch_size + b) * height_col) + h) * width_col + w;
      const int bp_dir = offset_c % 2;

      int j = (col_pos / width_col / height_col / batch_size) % kernel_w;
      int i =
          (col_pos / width_col / height_col / batch_size / kernel_w) % kernel_h;
      int w_out = col_pos % width_col;
      int h_out = (col_pos / width_col) % height_col;
      int w_in = w_out * stride_w - pad_w;
      int h_in = h_out * stride_h - pad_h;
      const int data_offset_h_ptr =
          (((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out);
      const int data_offset_w_ptr =
          (((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col +
           w_out);
      const int data_mask_hw_ptr =
          (((i * kernel_w + j) * height_col + h_out) * width_col + w_out);
      const scalar_t offset_h = data_offset_ptr[data_offset_h_ptr];
      const scalar_t offset_w = data_offset_ptr[data_offset_w_ptr];
      const scalar_t mask = data_mask_ptr[data_mask_hw_ptr];
      scalar_t inv_h = h_in + i * dilation_h + offset_h;
      scalar_t inv_w = w_in + j * dilation_w + offset_w;
      if (inv_h <= -1 || inv_w <= -1 || inv_h >= height || inv_w >= width) {
        inv_h = inv_w = -2;
      } else {
        mval += data_col_ptr[col_pos] *
            dmcn_im2col_bilinear(
                    data_im_ptr + cnt * height * width,
                    width,
                    height,
                    width,
                    inv_h,
                    inv_w);
      }
      const scalar_t weight = dmcn_get_coordinate_weight(
          inv_h,
          inv_w,
          height,
          width,
          data_im_ptr + cnt * height * width,
          width,
          bp_dir);
      val += weight * data_col_ptr[col_pos] * mask;
      cnt += 1;
    }
    // KERNEL_ASSIGN(grad_offset[index], offset_req, val);
    grad_offset[index] = val;
    if (offset_c % 2 == 0)
      // KERNEL_ASSIGN(grad_mask[(((b * deformable_group +
      // deformable_group_index) * kernel_h * kernel_w + offset_c / 2) *
      // height_col + h) * width_col + w], mask_req, mval);
      grad_mask
          [(((b * deformable_group + deformable_group_index) * kernel_h *
                 kernel_w +
             offset_c / 2) *
                height_col +
            h) *
               width_col +
           w] = mval;
  }
}


namespace detectron2 {

void modulated_deformable_im2col_cuda(
    const at::Tensor data_im,
    const at::Tensor data_offset,
    const at::Tensor data_mask,
    const int batch_size,
    const int channels,
    const int height_im,
    const int width_im,
    const int height_col,
    const int width_col,
    const int kernel_h,
    const int kenerl_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int deformable_group,
    at::Tensor data_col) {
  // num_axes should be smaller than block size
  const int channel_per_deformable_group = channels / deformable_group;
  const int num_kernels = channels * batch_size * height_col * width_col;

  at::cuda::CUDAGuard device_guard(data_im.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_im.scalar_type(), "modulated_deformable_im2col_gpu", ([&] {
        const scalar_t* data_im_ = data_im.data_ptr<scalar_t>();
        const scalar_t* data_offset_ = data_offset.data_ptr<scalar_t>();
        const scalar_t* data_mask_ = data_mask.data_ptr<scalar_t>();
        scalar_t* data_col_ = data_col.data_ptr<scalar_t>();

        modulated_deformable_im2col_gpu_kernel<<<
            GET_BLOCKS(num_kernels),
            CUDA_NUM_THREADS,
            0,
            stream>>>(
            num_kernels,
            data_im_,
            data_offset_,
            data_mask_,
            height_im,
            width_im,
            kernel_h,
            kenerl_w,
            pad_h,
            pad_w,
            stride_h,
            stride_w,
            dilation_h,
            dilation_w,
            channel_per_deformable_group,
            batch_size,
            channels,
            deformable_group,
            height_col,
            width_col,
            data_col_);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf(
        "error in modulated_deformable_im2col_cuda: %s\n",
        hipGetErrorString(err));
  }
}

void modulated_deformable_col2im_cuda(
    const at::Tensor data_col,
    const at::Tensor data_offset,
    const at::Tensor data_mask,
    const int batch_size,
    const int channels,
    const int height_im,
    const int width_im,
    const int height_col,
    const int width_col,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int deformable_group,
    at::Tensor grad_im) {
  const int channel_per_deformable_group = channels / deformable_group;
  const int num_kernels =
      channels * kernel_h * kernel_w * batch_size * height_col * width_col;

  at::cuda::CUDAGuard device_guard(data_col.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "modulated_deformable_col2im_gpu", ([&] {
        const scalar_t* data_col_ = data_col.data_ptr<scalar_t>();
        const scalar_t* data_offset_ = data_offset.data_ptr<scalar_t>();
        const scalar_t* data_mask_ = data_mask.data_ptr<scalar_t>();
        scalar_t* grad_im_ = grad_im.data_ptr<scalar_t>();

        modulated_deformable_col2im_gpu_kernel<<<
            GET_BLOCKS(num_kernels),
            CUDA_NUM_THREADS,
            0,
            stream>>>(
            num_kernels,
            data_col_,
            data_offset_,
            data_mask_,
            channels,
            height_im,
            width_im,
            kernel_h,
            kernel_w,
            pad_h,
            pad_w,
            stride_h,
            stride_w,
            dilation_h,
            dilation_w,
            channel_per_deformable_group,
            batch_size,
            deformable_group,
            height_col,
            width_col,
            grad_im_);
      }));

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf(
        "error in modulated_deformable_col2im_cuda: %s\n",
        hipGetErrorString(err));
  }
}

void modulated_deformable_col2im_coord_cuda(
    const at::Tensor data_col,
    const at::Tensor data_im,
    const at::Tensor data_offset,
    const at::Tensor data_mask,
    const int batch_size,
    const int channels,
    const int height_im,
    const int width_im,
    const int height_col,
    const int width_col,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int deformable_group,
    at::Tensor grad_offset,
    at::Tensor grad_mask) {
  const int num_kernels = batch_size * height_col * width_col * 2 * kernel_h *
      kernel_w * deformable_group;
  const int channel_per_deformable_group =
      channels * kernel_h * kernel_w / deformable_group;

  at::cuda::CUDAGuard device_guard(data_col.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "modulated_deformable_col2im_coord_gpu", ([&] {
        const scalar_t* data_col_ = data_col.data_ptr<scalar_t>();
        const scalar_t* data_im_ = data_im.data_ptr<scalar_t>();
        const scalar_t* data_offset_ = data_offset.data_ptr<scalar_t>();
        const scalar_t* data_mask_ = data_mask.data_ptr<scalar_t>();
        scalar_t* grad_offset_ = grad_offset.data_ptr<scalar_t>();
        scalar_t* grad_mask_ = grad_mask.data_ptr<scalar_t>();

        modulated_deformable_col2im_coord_gpu_kernel<<<
            GET_BLOCKS(num_kernels),
            CUDA_NUM_THREADS,
            0,
            stream>>>(
            num_kernels,
            data_col_,
            data_im_,
            data_offset_,
            data_mask_,
            channels,
            height_im,
            width_im,
            kernel_h,
            kernel_w,
            pad_h,
            pad_w,
            stride_h,
            stride_w,
            dilation_h,
            dilation_w,
            channel_per_deformable_group,
            batch_size,
            2 * kernel_h * kernel_w * deformable_group,
            deformable_group,
            height_col,
            width_col,
            grad_offset_,
            grad_mask_);
      }));
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf(
        "error in modulated_deformable_col2im_coord_cuda: %s\n",
        hipGetErrorString(err));
  }
}

} // namespace detectron2
