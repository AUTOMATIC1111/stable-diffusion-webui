// Copyright (c) Facebook, Inc. and its affiliates.


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace detectron2 {
int get_cudart_version() {
// Not a ROCM platform: Either HIP is not used, or
// it is used, but platform is not ROCM (i.e. it is CUDA)
#if !defined(__HIP_PLATFORM_HCC__)
  return CUDART_VERSION;
#else
  int version = 0;

#if HIP_VERSION_MAJOR != 0
  // Create a convention similar to that of CUDA, as assumed by other
  // parts of the code.

  version = HIP_VERSION_MINOR;
  version += (HIP_VERSION_MAJOR * 100);
#else
  hipRuntimeGetVersion(&version);
#endif
  return version;
#endif
}
} // namespace detectron2
