#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)

// Note: this implementation originates from the Caffe2 ROIAlignRotated Op
// and PyTorch ROIAlign (non-rotated) Op implementations.
// The key difference between this implementation and those ones is
// we don't do "legacy offset" in this version, as there aren't many previous
// works, if any, using the "legacy" ROIAlignRotated Op.
// This would make the interface a bit cleaner.

namespace detectron2 {

namespace {

template <typename T>
__device__ T bilinear_interpolate(
    const T* input,
    const int height,
    const int width,
    T y,
    T x) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    return 0;
  }

  if (y < 0) {
    y = 0;
  }

  if (x < 0) {
    x = 0;
  }

  int y_low = (int)y;
  int x_low = (int)x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = input[y_low * width + x_low];
  T v2 = input[y_low * width + x_high];
  T v3 = input[y_high * width + x_low];
  T v4 = input[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename T>
__device__ void bilinear_interpolate_gradient(
    const int height,
    const int width,
    T y,
    T x,
    T& w1,
    T& w2,
    T& w3,
    T& w4,
    int& x_low,
    int& x_high,
    int& y_low,
    int& y_high) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y < 0) {
    y = 0;
  }

  if (x < 0) {
    x = 0;
  }

  y_low = (int)y;
  x_low = (int)x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  // reference in forward
  // T v1 = input[y_low * width + x_low];
  // T v2 = input[y_low * width + x_high];
  // T v3 = input[y_high * width + x_low];
  // T v4 = input[y_high * width + x_high];
  // T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}

} // namespace

template <typename T>
__global__ void RoIAlignRotatedForward(
    const int nthreads,
    const T* input,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    const T* rois,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* current_roi = rois + n * 6;
    int roi_batch_ind = current_roi[0];

    // Do not use rounding; this implementation detail is critical
    // ROIAlignRotated supports align == true, i.e., continuous coordinate
    // by default, thus the 0.5 offset
    T offset = (T)0.5;
    T roi_center_w = current_roi[1] * spatial_scale - offset;
    T roi_center_h = current_roi[2] * spatial_scale - offset;
    T roi_width = current_roi[3] * spatial_scale;
    T roi_height = current_roi[4] * spatial_scale;
    T theta = current_roi[5] * M_PI / 180.0;
    T cos_theta = cos(theta);
    T sin_theta = sin(theta);

    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    const T* offset_input =
        input + (roi_batch_ind * channels + c) * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // roi_start_h and roi_start_w are computed wrt the center of RoI (x, y).
    // Appropriate translation needs to be applied after.
    T roi_start_h = -roi_height / 2.0;
    T roi_start_w = -roi_width / 2.0;

    // We do average (inte  gral) pooling inside a bin
    const T count = max(roi_bin_grid_h * roi_bin_grid_w, 1); // e.g. = 4

    T output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
    {
      const T yy = roi_start_h + ph * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T xx = roi_start_w + pw * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        // Rotate by theta around the center and translate
        T y = yy * cos_theta - xx * sin_theta + roi_center_h;
        T x = yy * sin_theta + xx * cos_theta + roi_center_w;

        T val = bilinear_interpolate(offset_input, height, width, y, x);
        output_val += val;
      }
    }
    output_val /= count;

    top_data[index] = output_val;
  }
}

template <typename T>
__global__ void RoIAlignRotatedBackwardFeature(
    const int nthreads,
    const T* top_diff,
    const int num_rois,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    T* bottom_diff,
    const T* rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* current_roi = rois + n * 6;
    int roi_batch_ind = current_roi[0];

    // Do not use rounding; this implementation detail is critical
    // ROIAlignRotated supports align == true, i.e., continuous coordinate
    // by default, thus the 0.5 offset
    T offset = (T)0.5;
    T roi_center_w = current_roi[1] * spatial_scale - offset;
    T roi_center_h = current_roi[2] * spatial_scale - offset;
    T roi_width = current_roi[3] * spatial_scale;
    T roi_height = current_roi[4] * spatial_scale;
    T theta = current_roi[5] * M_PI / 180.0;
    T cos_theta = cos(theta);
    T sin_theta = sin(theta);

    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    T* offset_bottom_diff =
        bottom_diff + (roi_batch_ind * channels + c) * height * width;

    int top_offset = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    const T top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // roi_start_h and roi_start_w are computed wrt the center of RoI (x, y).
    // Appropriate translation needs to be applied after.
    T roi_start_h = -roi_height / 2.0;
    T roi_start_w = -roi_width / 2.0;

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
    {
      const T yy = roi_start_h + ph * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T xx = roi_start_w + pw * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        // Rotate by theta around the center and translate
        T y = yy * cos_theta - xx * sin_theta + roi_center_h;
        T x = yy * sin_theta + xx * cos_theta + roi_center_w;

        T w1, w2, w3, w4;
        int x_low, x_high, y_low, y_high;

        bilinear_interpolate_gradient(
            height, width, y, x, w1, w2, w3, w4, x_low, x_high, y_low, y_high);

        T g1 = top_diff_this_bin * w1 / count;
        T g2 = top_diff_this_bin * w2 / count;
        T g3 = top_diff_this_bin * w3 / count;
        T g4 = top_diff_this_bin * w4 / count;

        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
          atomicAdd(
              offset_bottom_diff + y_low * width + x_low, static_cast<T>(g1));
          atomicAdd(
              offset_bottom_diff + y_low * width + x_high, static_cast<T>(g2));
          atomicAdd(
              offset_bottom_diff + y_high * width + x_low, static_cast<T>(g3));
          atomicAdd(
              offset_bottom_diff + y_high * width + x_high, static_cast<T>(g4));
        } // if
      } // ix
    } // iy
  } // CUDA_1D_KERNEL_LOOP
} // RoIAlignRotatedBackward

at::Tensor ROIAlignRotated_forward_cuda(
    const at::Tensor& input,
    const at::Tensor& rois,
    const float spatial_scale,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio) {
  AT_ASSERTM(input.device().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(rois.device().is_cuda(), "rois must be a CUDA tensor");
  at::TensorArg input_t{input, "input", 1}, rois_t{rois, "rois", 2};

  at::CheckedFrom c = "ROIAlignRotated_forward_cuda";
  at::checkAllSameGPU(c, {input_t, rois_t});
  at::checkAllSameType(c, {input_t, rois_t});
  at::cuda::CUDAGuard device_guard(input.device());

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto output = at::empty(
      {num_rois, channels, pooled_height, pooled_width}, input.options());
  auto output_size = num_rois * pooled_height * pooled_width * channels;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(
      at::cuda::ATenCeilDiv(
          static_cast<int64_t>(output_size), static_cast<int64_t>(512)),
      static_cast<int64_t>(4096)));
  dim3 block(512);

  if (output.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return output;
  }

  auto input_ = input.contiguous(), rois_ = rois.contiguous();
  AT_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "ROIAlignRotated_forward", [&] {
        RoIAlignRotatedForward<scalar_t><<<grid, block, 0, stream>>>(
            output_size,
            input_.data_ptr<scalar_t>(),
            spatial_scale,
            channels,
            height,
            width,
            pooled_height,
            pooled_width,
            sampling_ratio,
            rois_.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>());
      });
  hipDeviceSynchronize();
  AT_CUDA_CHECK(hipGetLastError());
  return output;
}

// TODO remove the dependency on input and use instead its sizes -> save memory
at::Tensor ROIAlignRotated_backward_cuda(
    const at::Tensor& grad,
    const at::Tensor& rois,
    const float spatial_scale,
    const int pooled_height,
    const int pooled_width,
    const int batch_size,
    const int channels,
    const int height,
    const int width,
    const int sampling_ratio) {
  AT_ASSERTM(grad.device().is_cuda(), "grad must be a CUDA tensor");
  AT_ASSERTM(rois.device().is_cuda(), "rois must be a CUDA tensor");

  at::TensorArg grad_t{grad, "grad", 1}, rois_t{rois, "rois", 2};
  at::CheckedFrom c = "ROIAlign_backward_cuda";
  at::checkAllSameGPU(c, {grad_t, rois_t});
  at::checkAllSameType(c, {grad_t, rois_t});
  at::cuda::CUDAGuard device_guard(grad.device());

  auto num_rois = rois.size(0);
  auto grad_input =
      at::zeros({batch_size, channels, height, width}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(
      at::cuda::ATenCeilDiv(
          static_cast<int64_t>(grad.numel()), static_cast<int64_t>(512)),
      static_cast<int64_t>(4096)));
  dim3 block(512);

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return grad_input;
  }

  auto grad_ = grad.contiguous(), rois_ = rois.contiguous();
  AT_DISPATCH_FLOATING_TYPES(
      grad.scalar_type(), "ROIAlignRotated_backward", [&] {
        RoIAlignRotatedBackwardFeature<scalar_t><<<grid, block, 0, stream>>>(
            grad.numel(),
            grad_.data_ptr<scalar_t>(),
            num_rois,
            spatial_scale,
            channels,
            height,
            width,
            pooled_height,
            pooled_width,
            sampling_ratio,
            grad_input.data_ptr<scalar_t>(),
            rois_.data_ptr<scalar_t>());
      });
  AT_CUDA_CHECK(hipGetLastError());
  return grad_input;
}

} // namespace detectron2
