#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#ifdef WITH_CUDA
#include "../box_iou_rotated/box_iou_rotated_utils.h"
#endif
// TODO avoid this when pytorch supports "same directory" hipification
#ifdef WITH_HIP
#include "box_iou_rotated/box_iou_rotated_utils.h"
#endif

using namespace detectron2;

namespace {
int const threadsPerBlock = sizeof(unsigned long long) * 8;
}

template <typename T>
__global__ void nms_rotated_cuda_kernel(
    const int n_boxes,
    const double iou_threshold,
    const T* dev_boxes,
    unsigned long long* dev_mask) {
  // nms_rotated_cuda_kernel is modified from torchvision's nms_cuda_kernel

  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
      min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
      min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  // Compared to nms_cuda_kernel, where each box is represented with 4 values
  // (x1, y1, x2, y2), each rotated box is represented with 5 values
  // (x_center, y_center, width, height, angle_degrees) here.
  __shared__ T block_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const T* cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      // Instead of devIoU used by original horizontal nms, here
      // we use the single_box_iou_rotated function from box_iou_rotated_utils.h
      if (single_box_iou_rotated<T>(cur_box, block_boxes + i * 5) >
          iou_threshold) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = at::cuda::ATenCeilDiv(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

namespace detectron2 {

at::Tensor nms_rotated_cuda(
    // input must be contiguous
    const at::Tensor& dets,
    const at::Tensor& scores,
    double iou_threshold) {
  // using scalar_t = float;
  AT_ASSERTM(dets.is_cuda(), "dets must be a CUDA tensor");
  AT_ASSERTM(scores.is_cuda(), "scores must be a CUDA tensor");
  at::cuda::CUDAGuard device_guard(dets.device());

  auto order_t = std::get<1>(scores.sort(0, /* descending=*/true));
  auto dets_sorted = dets.index_select(0, order_t);

  auto dets_num = dets.size(0);

  const int col_blocks =
      at::cuda::ATenCeilDiv(static_cast<int>(dets_num), threadsPerBlock);

  at::Tensor mask =
      at::empty({dets_num * col_blocks}, dets.options().dtype(at::kLong));

  dim3 blocks(col_blocks, col_blocks);
  dim3 threads(threadsPerBlock);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES(
      dets_sorted.scalar_type(), "nms_rotated_kernel_cuda", [&] {
        nms_rotated_cuda_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
            dets_num,
            iou_threshold,
            dets_sorted.data_ptr<scalar_t>(),
            (unsigned long long*)mask.data_ptr<int64_t>());
      });

  at::Tensor mask_cpu = mask.to(at::kCPU);
  unsigned long long* mask_host =
      (unsigned long long*)mask_cpu.data_ptr<int64_t>();

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  at::Tensor keep =
      at::empty({dets_num}, dets.options().dtype(at::kLong).device(at::kCPU));
  int64_t* keep_out = keep.data_ptr<int64_t>();

  int num_to_keep = 0;
  for (int i = 0; i < dets_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long* p = mask_host + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }

  AT_CUDA_CHECK(hipGetLastError());
  return order_t.index(
      {keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep)
           .to(order_t.device(), keep.scalar_type())});
}

} // namespace detectron2
